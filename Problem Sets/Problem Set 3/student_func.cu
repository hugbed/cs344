#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

#include <stdio.h>
#include <stdlib.h>

#define OP_MAX 3
#define OP_MIN 2

#define THREADBLOCK_SIZE 256

typedef unsigned int uint;

__global__ 
void reduce(float* d_out,
			const float* const d_in,
			const unsigned int op)
{
	extern __shared__ float stmp[];

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	// load data into shared memory (one value per thread)
	int tid = threadIdx.x;
	stmp[tid] = d_in[i];
	__syncthreads();

	for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1) {
		if (tid < s) {
			if (op == OP_MAX) {
				stmp[tid] = max(stmp[tid], stmp[tid + s]);
			} else if (op == OP_MIN){
				stmp[tid] = min(stmp[tid], stmp[tid + s]);
			}
		}
		__syncthreads();
	}

	if (tid == 0) {
		d_out[blockIdx.x] = stmp[0];
	}
}

__global__
void histogram(uint* d_bins,
			   const float* const d_in,
			   const float lumMin,
			   const float lumMax,
			   const size_t numBins)
{
	extern __shared__ float s_in[];

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	// load data into shared memory (one value per thread)
	int tid = threadIdx.x;
	s_in[tid] = d_in[i];
	__syncthreads();

	// could initialize to 0 some way here
	// here <--

	// could build histogram in shared memory, then
	// merge it to global histogram

	float lumRange = lumMax - lumMin;
	unsigned int binIdx = (s_in[tid] - lumMin) / lumRange * numBins;

	// could probably be faster somehow else
	atomicAdd(&(d_bins[binIdx]), 1);
}


////////////////////////////////////////////////////////////////////////////////
// Basic scan codelets
////////////////////////////////////////////////////////////////////////////////
//Naive inclusive scan: O(N * log2(N)) operations
//Allocate 2 * 'size' local memory, initialize the first half
//with 'size' zeros avoiding if(pos >= offset) condition evaluation
//and saving instructions
inline __device__ uint scan1Inclusive(uint idata, volatile uint *s_Data, uint size)
{
    uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
    s_Data[pos] = 0;
    pos += size;
    s_Data[pos] = idata;

    for (uint offset = 1; offset < size; offset <<= 1)
    {
        __syncthreads();
        uint t = s_Data[pos] + s_Data[pos - offset];
        __syncthreads();
        s_Data[pos] = t;
    }

    return s_Data[pos];
}

inline __device__ uint scan1Exclusive(uint idata, volatile uint *s_Data, uint size)
{
    return scan1Inclusive(idata, s_Data, size) - idata;
}


inline __device__ uint4 scan4Inclusive(uint4 idata4, volatile uint *s_Data, uint size)
{
    //Level-0 inclusive scan
    idata4.y += idata4.x;
    idata4.z += idata4.y;
    idata4.w += idata4.z;

    //Level-1 exclusive scan
    uint oval = scan1Exclusive(idata4.w, s_Data, size / 4);

    idata4.x += oval;
    idata4.y += oval;
    idata4.z += oval;
    idata4.w += oval;

    return idata4;
}

//Exclusive vector scan: the array to be scanned is stored
//in local thread memory scope as uint4
inline __device__ uint4 scan4Exclusive(uint4 idata4, volatile uint *s_Data, uint size)
{
    uint4 odata4 = scan4Inclusive(idata4, s_Data, size);
    odata4.x -= idata4.x;
    odata4.y -= idata4.y;
    odata4.z -= idata4.z;
    odata4.w -= idata4.w;
    return odata4;
}

////////////////////////////////////////////////////////////////////////////////
// Scan kernels
////////////////////////////////////////////////////////////////////////////////
__global__ void scanExclusiveShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint size
)
{
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load data
    uint4 idata4 = d_Src[pos];

    //Calculate exclusive scan
    uint4 odata4 = scan4Exclusive(idata4, s_Data, size);

    //Write back
    d_Dst[pos] = odata4;
}

////////////////////////////////////////////////////////////////////////////////
// Interface function
////////////////////////////////////////////////////////////////////////////////
//Derived as 32768 (max power-of-two gridDim.x) * 4 * THREADBLOCK_SIZE
//Due to scanExclusiveShared<<<>>>() 1D block addressing
extern "C" const uint MAX_BATCH_ELEMENTS = 64 * 1048576;
extern "C" const uint MIN_SHORT_ARRAY_SIZE = 4;
extern "C" const uint MAX_SHORT_ARRAY_SIZE = 4 * THREADBLOCK_SIZE;

static uint factorRadix2(uint &log2L, uint L)
{
    if (!L)
    {
        log2L = 0;
        return 0;
    }
    else
    {
        for (log2L = 0; (L & 1) == 0; L >>= 1, log2L++);

        return L;
    }
}

static uint iDivUp(uint dividend, uint divisor)
{
    return ((dividend % divisor) == 0) ? (dividend / divisor) : (dividend / divisor + 1);
}

extern "C" size_t scanExclusiveShort(
    uint *d_Dst,
    uint *d_Src,
    uint batchSize,
    uint arrayLength
)
{
    //Check power-of-two factorization
    uint log2L;
    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
    assert(factorizationRemainder == 1);

    //Check supported size range
    assert((arrayLength >= MIN_SHORT_ARRAY_SIZE) && (arrayLength <= MAX_SHORT_ARRAY_SIZE));

    //Check total batch size limit
    assert((batchSize * arrayLength) <= MAX_BATCH_ELEMENTS);

    //Check all threadblocks to be fully packed with data
    assert((batchSize * arrayLength) % (4 * THREADBLOCK_SIZE) == 0);

    scanExclusiveShared<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint4 *)d_Src,
        arrayLength
    );

    return THREADBLOCK_SIZE;
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
	
	// 1)
	size_t n = numRows * numCols;
	int blockSize = 1024;
	int gridSize = n / blockSize;
	
	float *d_intermediate, *d_out;
	checkCudaErrors(hipMalloc(&d_intermediate, n * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_out, sizeof(float)));
	
	float h_out;
	reduce<<<gridSize, blockSize, blockSize * sizeof(float)>>>(d_intermediate, d_logLuminance, OP_MAX);
	reduce<<<gridSize, blockSize, blockSize * sizeof(float)>>>(d_out, d_intermediate, OP_MAX);
	checkCudaErrors(hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost));
	max_logLum = h_out;
	
	reduce<<<gridSize, blockSize, blockSize * sizeof(float)>>>(d_intermediate, d_logLuminance, OP_MIN);
	reduce<<<gridSize, blockSize, blockSize * sizeof(float)>>>(d_out, d_intermediate, OP_MIN);
	checkCudaErrors(hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost));
	min_logLum = h_out;

	checkCudaErrors(hipFree(d_intermediate));
	checkCudaErrors(hipFree(d_out));

	uint *d_bins;
	checkCudaErrors(hipMalloc(&d_bins, numBins * sizeof(uint)));
	checkCudaErrors(hipMemset(d_bins, 0, numBins * sizeof(uint)));
	histogram<<<gridSize, blockSize, blockSize * sizeof(float)>>>(d_bins, d_logLuminance, min_logLum, max_logLum, numBins);

	scanExclusiveShort(d_cdf, d_bins, 1, numBins);
}
